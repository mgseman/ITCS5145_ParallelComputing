
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void VecAdd(float *d_in1, float *d_in2, float *d_out)
{
  int i = threadIdx.x;
  d_out[i] = d_in1[i] + d_in2[i];
}

int main (int argc, char* argv[]) {
  //TODO: add usage
  
  int n = atoi(argv[1]); //TODO: atoi is an unsafe function

  float* in1 = new float[n];
  float* in2 = new float[n];
  float* out = new float[n];
  
  
  for (int i=0; i<n; ++i) {
    in1[i] = i;
    in2[i] = 100+i;
  }


  // Start Timer
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  //gpu stuff goes here.

  // Declare and allocate GPU memory
  float *d_in1;
  float *d_in2;
  float *d_out;
  hipMalloc((void **) &d_in1, n);
  hipMalloc((void **) &d_in2, n);
  hipMalloc((void **) &d_out, n);

  // Copy input arrays onto GPU memory
  hipMemcpy(d_in1, in1, n, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2, n, hipMemcpyHostToDevice);

  // Perform vector addition on GPU
  VecAdd<<<1, n>>>(d_in1, d_in2, d_out);

  // Copy output of vector addition onto CPU memory
  hipMemcpy(d_out, out, n, hipMemcpyHostToDevice);
  
  // Free GPU memory
  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin);

  std::cout<<n<<" "<<totaltime.count()<<std::endl;

  delete[] in1;
  delete[] in2;
  delete[] out;

  return 0;
}
