
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void VecAdd(float *d_in1, float *d_in2, float *d_out)
{
  int i = threadIdx.x;
  d_out[i] = d_in1[i] + d_in2[i];
}

int main (int argc, char* argv[]) {
  //TODO: add usage
  
  int n = atoi(argv[1]); //TODO: atoi is an unsafe function

  float* in1 = new float[n];
  float* in2 = new float[n];
  float* out = new float[n];
  
  
  for (int i=0; i<n; ++i) {
    in1[i] = i;
    in2[i] = 100+i;
  }


  // Start Timer
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  //gpu stuff goes here.

  // Declare and allocate GPU memory
  float *d_in1;
  float *d_in2;
  float *d_out;
  hipMalloc((void **) &d_in1, n*sizeof(float));
  hipMalloc((void **) &d_in2, n*sizeof(float));
  hipMalloc((void **) &d_out, n*sizeof(float));

  // Copy input arrays onto GPU memory
  hipMemcpy(d_in1, in1, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2, n*sizeof(float), hipMemcpyHostToDevice);

  // Perform vector addition on GPU
  VecAdd<<<1, n*sizeof(float)>>>(d_in1, d_in2, d_out);

  // Copy output of vector addition onto CPU memory
  hipMemcpy(out, d_out, n*sizeof(float), hipMemcpyDeviceToHost);
  
  // Free GPU memory
  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin);

  std::cout<<n<<" "<<totaltime.count()<<std::endl;

  delete[] in1;
  delete[] in2;
  delete[] out;

  return 0;
}
