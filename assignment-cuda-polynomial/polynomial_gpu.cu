
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void polynomial_expansion (float* poly, int degree,
			   int n, float* array) {
  //TODO: Write code to use the GPU here!
  //code should write the output back to array

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) 
  {
    float out = 0.;
    float xtothepowerof = 1.;
    for (int i=0; i<=degree; ++i) {
      out += xtothepowerof*poly[i];
      xtothepowerof *= array[idx];
    }
    array[idx] = out;
  }
}


int main (int argc, char* argv[]) {
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  float *d_poly;
  float *d_arr;
  hipMalloc((void **) &d_poly, (degree+1)*sizeof(float));
  hipMalloc((void **) &d_arr, n*sizeof(float));

  // Copy input arrays onto GPU memory
  hipMemcpy(d_poly, poly, (degree+1)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_arr, array, n*sizeof(float), hipMemcpyHostToDevice);

  // Perform Polynomial expansion on GPU
  for (int iter = 0; iter<nbiter; ++iter) {
    polynomial_expansion<<<(n+255)/256, 256>>>(d_poly, degree, n, d_arr);
  }
  
  // Send array back to CPU memory
  hipMemcpy(array, d_arr, n*sizeof(float), hipMemcpyDeviceToHost);

  // Free CUDA memory
  hipFree(d_arr);
  hipFree(d_poly);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
	      ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;

  return 0;
}
